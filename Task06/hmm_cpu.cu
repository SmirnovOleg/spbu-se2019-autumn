#include <iostream>
#include <random>
#include <iomanip>
#include <cstring>
#include "hmm.hpp"

using namespace std;


double HiddenMarkovModel::run_forward_algo() {
    // alpha[t][i] is the TOTAL probability
    //      of all observations up to time [t] in the hidden state [i]
    for (int i = 0; i < n_states; ++i) {
        alpha[0][i] = initial_distribution[i] * emissions[i][observations[0]];
    }
    for (int t = 1; t < n_observations; ++t) {
        for (int i = 0; i < n_states; ++i) {
            alpha[t][i] = 0.0;
            for (int j = 0; j < n_states; ++j) {
                alpha[t][i] += alpha[t - 1][j] * transitions[j][i] * emissions[i][observations[t]];
            }
        }
    }
    double likelihood = 0.0;
    for (int i = 0; i < n_states; ++i) {
        likelihood += alpha[n_observations - 1][i];
    }
    return likelihood;
}


void HiddenMarkovModel::run_viterbi_algo(int *hidden_states_idxs) {
    // alpha[t][i] is the MOST LIKELY probability 
    //      of all observations up to time [t] in the hidden state [i]
    for (int i = 0; i < n_states; ++i) {
        alpha[0][i] = initial_distribution[i] * emissions[i][observations[0]];
    }
    int **backtrack = new int *[n_observations];
    for (int i = 0; i < n_observations; i++) {
        backtrack[i] = new int [n_states];
    }
    for (int t = 1; t <= n_observations; ++t) {
        for (int i = 0; i < n_states; ++i) {
            if (t != n_observations) {
                alpha[t][i] = -1;
                for (int j = 0; j < n_states; ++j) {
                    double prob = transitions[j][i] * emissions[i][observations[t]];
                    if (alpha[t][i] < alpha[t - 1][j] * prob) {
                        alpha[t][i] = alpha[t - 1][j] * prob;
                    }
                }
            }
            double most_likely_prev_prob = -1;
            for (int j = 0; j < n_states; ++j) {
                if (most_likely_prev_prob < alpha[t - 1][j] * transitions[j][i]) {
                    most_likely_prev_prob = alpha[t - 1][j] * transitions[j][i];
                    backtrack[t - 1][i] = j;
                }
            }
        }
    }
    // Calculate the most likely last hidden state
    double max_likelihood = -1;
    for (int i = 0; i < n_states; ++i) {
        if (alpha[n_observations - 1][i] > max_likelihood) {
            max_likelihood = alpha[n_observations - 1][i];
            hidden_states_idxs[n_observations - 1] = i;
        }
    }
    // Go backtracking
    for (int i = n_observations - 2; i >= 0; --i) {
        int next_state = hidden_states_idxs[i + 1];
        hidden_states_idxs[i] = backtrack[i + 1][next_state];
    }
}


void HiddenMarkovModel::run_baum_welch_algo(int epochs) {
    generate_random_parameters();
    // Allocate memory
    auto **gamma = new double *[n_observations];
    auto **xi = new double *[n_states];
    for (int i = 0; i < n_observations; ++i) {
        gamma[i] = new double[n_states];
    }
    for (int i = 0; i < n_states; ++i) {
        xi[i] = new double[n_states];
    }
    // Fit the model
    for (int epoch = 0; epoch < epochs; ++epoch) {
        double likelihood = run_forward_algo();
        if (epoch % 10 == 0) {
            cout << "Epoch: " << epoch << " | Likelihood: " << likelihood << endl;
        }
        // Calculate probabilities: beta, gamma and xi
        for (int i = 0; i < n_states; ++i) {
            beta[n_observations - 1][i] = 1;
            gamma[n_observations - 1][i] = (alpha[n_observations - 1][i] * beta[n_observations - 1][i]) / likelihood;
            memset(xi[i], 0.0, n_states * sizeof(double));
        }
        for (int t = n_observations - 2; t >= 0; --t) {
            for (int i = 0; i < n_states; ++i) {
                beta[t][i] = 0.0;
                for (int j = 0; j < n_states; ++j) {
                    double prob = transitions[i][j] * emissions[j][observations[t + 1]];
                    beta[t][i] += beta[t + 1][j] * prob;
                    xi[i][j] += (alpha[t][i] * prob * beta[t + 1][j]) / likelihood;
                }
                gamma[t][i] = (alpha[t][i] * beta[t][i]) / likelihood;
            }
        }
        // Optimize parameters
        for (int i = 0; i < n_states; ++i) {
            double occupation_prob = 0.0;
            for (int t = 0; t < n_observations - 1; ++t) {
                occupation_prob += gamma[t][i];
            }
            for (int j = 0; j < n_states; ++j) {
                transitions[i][j] = xi[i][j] / occupation_prob;
            }
            double total_occupation_prob = occupation_prob + gamma[n_observations - 1][i];
            for (int j = 0; j < n_observations; ++j) {
                double sum = 0.0;
                for (int t = 0; t < n_observations; ++t) {
                    sum += (observations[t] == observations[j]) ? gamma[t][i] : 0;
                }
                emissions[i][observations[j]] = sum / total_occupation_prob;
            }
            initial_distribution[i] = gamma[0][i];
        }
    }
    // Free memory
    for (int i = 0; i < n_observations; ++i) {
        delete gamma[i];
    }
    delete[] gamma;
    for (int i = 0; i < n_states; ++i) {
        delete xi[i];
    }
    delete[] xi;
}
