#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <iomanip>
#include <cstring>
#include "hmm.hpp"

using namespace std;

const int THREADS_PER_BLOCK = 1024;


__global__
void make_forward_step(double *alpha, double *transitions, double *emissions, int *observations,
                       int t, int n_states, int observations_dict_size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_states) {
        alpha[t * n_states + i] = 0.0;
        for (int j = 0; j < n_states; ++j) {
            alpha[t * n_states + i] += alpha[(t - 1) * n_states + j] 
                                        * transitions[j * n_states + i] 
                                        * emissions[i * observations_dict_size + observations[t]];
        }
    }
}


template <typename T>
T *allocate_matrix_on_device(int rows, int cols){
    T *matrix;
    hipMalloc(&matrix, sizeof(T) * rows * cols);
    return matrix;
}


template <typename T>
void copy_matrix_to_device(T *d_matrix, T **matrix, int rows, int cols) {
    T **temp_matrix = new T *[rows * cols];
    for (int i = 0; i < rows; ++i)
        memcpy(temp_matrix + i * cols, matrix[i], cols * sizeof(T));
    hipMemcpy(d_matrix, temp_matrix, rows * cols * sizeof(T), hipMemcpyHostToDevice);
    delete[] temp_matrix;
}


template <typename T>
void copy_matrix_from_device(T **matrix, T *d_matrix, int rows, int cols) {
    T **temp_matrix = new T *[rows * cols];
    hipMemcpy(temp_matrix, d_matrix, rows * cols * sizeof(T), hipMemcpyDeviceToHost);
    for (int i = 0; i < rows; ++i) {
        memcpy(matrix[i], temp_matrix + i * cols, cols * sizeof(T));
    }
    delete[] temp_matrix;
}


double HiddenMarkovModel::run_GPU_forward_algo() {
    for (int i = 0; i < n_states; ++i) {
        alpha[0][i] = initial_distribution[i] * emissions[i][observations[0]];
    }

    double *d_alpha = allocate_matrix_on_device<double>(n_observations, n_states);
    double *d_transitions = allocate_matrix_on_device<double>(n_states, n_states);
    double *d_emissions = allocate_matrix_on_device<double>(n_states, observations_dict_size);
    int *d_observations;
    hipMalloc(&d_observations, n_observations * sizeof(int));

    copy_matrix_to_device<double>(d_alpha, alpha, n_observations, n_states);
    copy_matrix_to_device<double>(d_transitions, transitions, n_states, n_states);
    copy_matrix_to_device<double>(d_emissions, emissions, n_states, observations_dict_size);
    hipMemcpy(d_observations, observations, n_observations * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    for (int t = 1; t < n_observations; ++t) {
        make_forward_step<<<n_states/THREADS_PER_BLOCK+1, THREADS_PER_BLOCK>>>(
                d_alpha, d_transitions, d_emissions, d_observations,
                t, n_states, observations_dict_size);
        hipDeviceSynchronize();
    }

    copy_matrix_from_device<double>(alpha, d_alpha, n_observations, n_states);

    double likelihood = 0.0;
    for (int i = 0; i < n_states; ++i) {
        likelihood += alpha[n_observations - 1][i];
    }

    hipFree(d_alpha);
    hipFree(d_transitions);
    hipFree(d_emissions);
    hipFree(d_observations);

    return likelihood;
}


__global__
void make_viterbi_forward_step(double *alpha, double *transitions, double *emissions, int *observations,
                                int t, int n_states, int observations_dict_size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_states) {
        alpha[t * n_states + i] = -1;
        for (int j = 0; j < n_states; ++j) {
            double prob = transitions[j * n_states + i] * emissions[i * observations_dict_size + observations[t]];
            if (alpha[t * n_states + i] < alpha[(t - 1) * n_states + j] * prob) {
                alpha[t * n_states + i] = alpha[(t - 1) * n_states + j] * prob;
            }
        }
    }
}


__global__
void make_viterbi_backward_step(double *alpha, double *transitions, double *backtrack, int t, int n_states) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_states) {
        double most_likely_prev_prob = -1;
        for (int j = 0; j < n_states; ++j) {
            if (most_likely_prev_prob < alpha[t * n_states + j] * transitions[j * n_states + i]) {
                most_likely_prev_prob = alpha[t * n_states + j] * transitions[j * n_states + i];
                backtrack[t * n_states + i] = j;
            }
        }
    }
}


void HiddenMarkovModel::run_GPU_viterbi_algo(int *hidden_states_idxs) {
    for (int i = 0; i < n_states; ++i) {
        alpha[0][i] = initial_distribution[i] * emissions[i][observations[0]];
    }
    double **backtrack = new double *[n_observations];
    for (int i = 0; i < n_observations; i++) {
        backtrack[i] = new double [n_states];
    }

    double *d_alpha = allocate_matrix_on_device<double>(n_observations, n_states);
    double *d_transitions = allocate_matrix_on_device<double>(n_states, n_states);
    double *d_emissions = allocate_matrix_on_device<double>(n_states, observations_dict_size);
    double *d_backtrack = allocate_matrix_on_device<double>(n_observations, n_states);
    int *d_observations;
    hipMalloc(&d_observations, n_observations * sizeof(int));

    copy_matrix_to_device<double>(d_alpha, alpha, n_observations, n_states);
    copy_matrix_to_device<double>(d_transitions, transitions, n_states, n_states);
    copy_matrix_to_device<double>(d_emissions, emissions, n_states, observations_dict_size);
    copy_matrix_to_device<double>(d_backtrack, backtrack, n_observations, n_states);
    hipMemcpy(d_observations, observations, n_observations * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    for (int t = 1; t < n_observations; ++t) {
        make_viterbi_forward_step<<<n_states/THREADS_PER_BLOCK+1, THREADS_PER_BLOCK>>>(
            d_alpha, d_transitions, d_emissions, d_observations, 
            t, n_states, observations_dict_size);
        hipDeviceSynchronize();
    }
    for (int t = 0; t < n_observations; ++t) {
        make_viterbi_backward_step<<<n_states/THREADS_PER_BLOCK+1, THREADS_PER_BLOCK>>>(
            d_alpha, d_transitions, d_backtrack, t, n_states);
    }
    
    copy_matrix_from_device<double>(alpha, d_alpha, n_observations, n_states);
    copy_matrix_from_device<double>(backtrack, d_backtrack, n_observations, n_states);
    hipFree(d_backtrack);
    hipFree(d_alpha);
    hipFree(d_transitions);
    hipFree(d_emissions);
    hipFree(d_observations);

    // Calculate the most likely last hidden state
    double max_likelihood = -1;
    for (int i = 0; i < n_states; ++i) {
        if (alpha[n_observations - 1][i] > max_likelihood) {
            max_likelihood = alpha[n_observations - 1][i];
            hidden_states_idxs[n_observations - 1] = i;
        }
    }
    // Go backtracking
    for (int i = n_observations - 2; i >= 0; --i) {
        int next_state = hidden_states_idxs[i + 1];
        hidden_states_idxs[i] = (int) backtrack[i + 1][next_state];
    }
    // Free memory
    for (int i = 0; i < n_observations; ++i) {
        delete backtrack[i];
    }
    delete[] backtrack;
}


void HiddenMarkovModel::run_GPU_baum_welch_algo(int epochs) {
    generate_random_parameters();
    // Allocate memory
    auto **gamma = new double *[n_observations];
    auto **xi = new double *[n_states];
    for (int i = 0; i < n_observations; ++i) {
        gamma[i] = new double[n_states];
    }
    for (int i = 0; i < n_states; ++i) {
        xi[i] = new double[n_states];
    }
    // Fit the model
    for (int epoch = 0; epoch < epochs; ++epoch) {
        double likelihood = run_GPU_forward_algo();
        if (epoch % 10 == 0) {
            cout << "Epoch: " << epoch << " | Likelihood: " << likelihood << endl;
        }
        // Calculate probabilities: beta, gamma and xi
        for (int i = 0; i < n_states; ++i) {
            beta[n_observations - 1][i] = 1;
            gamma[n_observations - 1][i] = (alpha[n_observations - 1][i] * beta[n_observations - 1][i]) / likelihood;
            memset(xi[i], 0.0, n_states * sizeof(double));
        }
        for (int t = n_observations - 2; t >= 0; --t) {
            for (int i = 0; i < n_states; ++i) {
                beta[t][i] = 0.0;
                for (int j = 0; j < n_states; ++j) {
                    double prob = transitions[i][j] * emissions[j][observations[t + 1]];
                    beta[t][i] += beta[t + 1][j] * prob;
                    xi[i][j] += (alpha[t][i] * prob * beta[t + 1][j]) / likelihood;
                }
                gamma[t][i] = (alpha[t][i] * beta[t][i]) / likelihood;
            }
        }
        // Optimize parameters
        for (int i = 0; i < n_states; ++i) {
            double occupation_prob = 0.0;
            for (int t = 0; t < n_observations - 1; ++t) {
                occupation_prob += gamma[t][i];
            }
            for (int j = 0; j < n_states; ++j) {
                transitions[i][j] = xi[i][j] / occupation_prob;
            }
            double total_occupation_prob = occupation_prob + gamma[n_observations - 1][i];
            for (int j = 0; j < n_observations; ++j) {
                double sum = 0.0;
                for (int t = 0; t < n_observations; ++t) {
                    sum += (observations[t] == observations[j]) ? gamma[t][i] : 0;
                }
                emissions[i][observations[j]] = sum / total_occupation_prob;
            }
            initial_distribution[i] = gamma[0][i];
        }
    }
    // Free memory
    for (int i = 0; i < n_observations; ++i) {
        delete gamma[i];
    }
    delete[] gamma;
    for (int i = 0; i < n_states; ++i) {
        delete xi[i];
    }
    delete[] xi;
}
